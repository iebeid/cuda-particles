#include "hip/hip_runtime.h"
﻿
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <iostream>
#include <cmath>
#include <string>
#include <fstream>
#include <map>
#include <future>

#include <glm/glm.hpp>
#include <glm/vec3.hpp>
#include <glm/vec4.hpp>
#include <glm/mat4x4.hpp>
#include <glm/gtc/matrix_transform.hpp> 
#include <glm/gtc/type_ptr.hpp>

#include <hip/hip_runtime.h>
#include <>

#include <GL/glew.h>
#include <GLFW/glfw3.h>

using namespace std;

float y_rotation = 0.0f;
float angle = 0.0f;
int n = 300000;

class v3
{
public:
	float x;
	float y;
	float z;

	v3(){ randomize(); }
	v3(float xIn, float yIn, float zIn) : x(xIn), y(yIn), z(zIn)
	{}
	void randomize(){
		x = (float)rand() / (float)RAND_MAX;
		y = (float)rand() / (float)RAND_MAX;
		z = (float)rand() / (float)RAND_MAX;
	}
	__host__ __device__ void normalize(){
		float t = sqrt(x*x + y*y + z*z);
		x /= t;
		y /= t;
		z /= t;
	}
	__host__ __device__ void scramble(){
		float tx = 0.317f*(x + 1.0) + y + z * x * x + y + z;
		float ty = 0.619f*(y + 1.0) + y * y + x * y * z + y + x;
		float tz = 0.124f*(z + 1.0) + z * y + x * y * z + y + x;

		//float tx = x;
		//float ty = y;
		//float tz = z;


		x = tx;
		y = ty;
		z = tz;
	}

};

class particle
{
public:
	v3 position;
	v3 velocity;
	v3 totalDistance;
	float life;

public:
	particle() : position(), velocity(), totalDistance(0, 0, 0), life()
	{}
	__host__ __device__ void advance(float d){
		velocity.normalize();
		float dx = d * velocity.x * velocity.x;
		position.x += dx;
		totalDistance.x += dx;
		float dy = d * velocity.y * velocity.y;
		position.y += dy;
		totalDistance.y += dy;
		float dz = d * velocity.z * velocity.z;
		position.z += dz;
		totalDistance.z += dz;
		life -= d;
		velocity.scramble();
	}

	const v3& getTotalDistance() const{
		return totalDistance;
	}

};

__global__ void advanceParticles(float dt, particle * pArray, int nParticles)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if (idx < nParticles)
	{
		pArray[idx].advance(dt);
	}
}

void controls(GLFWwindow* window, int key, int scancode, int action, int mods)
{
	if (action == GLFW_PRESS)
		if (key == GLFW_KEY_ESCAPE)
			glfwSetWindowShouldClose(window, GL_TRUE);
}

void mouse_button_callback(GLFWwindow* window, int button, int action, int mods)
{
	if (button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_PRESS)
		angle = angle + 0.1f;
}

GLFWwindow* initWindow(const int resX, const int resY)
{
	if (!glfwInit())
	{
		fprintf(stderr, "Failed to initialize GLFW\n");
		return NULL;
	}
	glfwWindowHint(GLFW_SAMPLES, 4);
	GLFWwindow* window = glfwCreateWindow(resX, resY, "Render Cuda Particles", NULL, NULL);
	if (window == NULL)
	{
		fprintf(stderr, "Failed to open GLFW window.\n");
		glfwTerminate();
		return NULL;
	}
	glfwMakeContextCurrent(window);
	glfwSetKeyCallback(window, controls);
	glfwSetMouseButtonCallback(window, mouse_button_callback);
	glewExperimental = GL_TRUE;
	glewInit();
	printf("Renderer: %s\n", glGetString(GL_RENDERER));
	printf("OpenGL version supported %s\n", glGetString(GL_VERSION));
	glEnable(GL_DEPTH_TEST);
	glDepthMask(GL_TRUE);
	glDepthFunc(GL_LEQUAL);
	glCullFace(GL_BACK);
	return window;
}

void display(GLFWwindow* window)
{
	particle * pArray = new particle[n];
	particle * devPArray = NULL;
	hipMalloc(&devPArray, n*sizeof(particle));
	hipMemcpy(devPArray, pArray, n*sizeof(particle), hipMemcpyHostToDevice);

	std::string vertex_line, vertex_text;
	std::ifstream vertex_in("vertex.vert");
	while (std::getline(vertex_in, vertex_line))
	{
		vertex_text += vertex_line + "\n";
	}
	
	std::string frag_line, frag_text;
	std::ifstream frag_in("fragment.frag");
	while (std::getline(frag_in, frag_line))
	{
		frag_text += frag_line + "\n";
	}

	const char* vertex_data = vertex_text.c_str();
	const char* fragment_data = frag_text.c_str();
	const char* vertex_shader = vertex_data;
	const char* fragment_shader = fragment_data;

	GLfloat *colors = new GLfloat[n * 3];
	int j = 0;
	for (int i = 0; i<n; i = i + 3)
	{
		colors[i] = ((float)rand() / (RAND_MAX)) + 1;
		colors[i + 1] = ((float)rand() / (RAND_MAX)) + 1;
		colors[i + 2] = ((float)rand() / (RAND_MAX)) + 1;
	}
	GLfloat *vertices = new GLfloat[n * 3];
	while (!glfwWindowShouldClose(window))
	{
		//Calulations
		float dt = (float)rand() / (float)RAND_MAX;
		advanceParticles <<< 1 + n / 256, 256 >>>(dt, devPArray, n);
		hipDeviceSynchronize();
		hipMemcpy(pArray, devPArray, n * sizeof(particle), hipMemcpyDeviceToHost);
		
		//GLfloat *colors = new GLfloat[n * 3];
		int j = 0;
		for (int i = 0; i<n; i = i + 3)
		{
			v3 pos = pArray[j].position;
			float vertex_magnitude = sqrt(pow(pos.x, 2) + pow(pos.y, 2) + pow(pos.z, 2));
			vertices[i] = pos.x / vertex_magnitude;
			vertices[i + 1] = pos.y / vertex_magnitude;
			vertices[i + 2] = pos.z / vertex_magnitude;

			//colors[i] = ((float)rand() / (RAND_MAX)) + 1;
			//colors[i + 1] = ((float)rand() / (RAND_MAX)) + 1;
			//colors[i + 2] = ((float)rand() / (RAND_MAX)) + 1;
			j++;
		}

		// Scale to window size
		GLint windowWidth, windowHeight;
		glfwGetWindowSize(window, &windowWidth, &windowHeight);
		glViewport(0, 0, windowWidth, windowHeight);

		glClearColor(0.1, 0.1, 0.1, 1.0);
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glMatrixMode(GL_PROJECTION_MATRIX);
		glm::mat4 Projection = glm::perspective(60.0f, (float)windowWidth / (float)windowHeight, 0.1f, 1000.f);

		glMatrixMode(GL_MODELVIEW_MATRIX);

		glRotatef(angle, 0.0f, 1.0f, 0.0f);

		GLuint vboId;
		GLuint cboId;

		glGenBuffers(1, &vboId);
		glBindBuffer(GL_ARRAY_BUFFER, vboId);
		glBufferData(GL_ARRAY_BUFFER, 3 * n * sizeof(GLfloat), 0, GL_STATIC_DRAW);

		glGenBuffers(1, &cboId);
		glBindBuffer(GL_ARRAY_BUFFER, cboId);
		glBufferData(GL_ARRAY_BUFFER, 3 * n * sizeof(GLfloat), 0, GL_STATIC_DRAW);

		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_COLOR_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, vboId);
		glBufferData(GL_ARRAY_BUFFER, 3 * n * sizeof(GLfloat), vertices, GL_STATIC_DRAW);
		glVertexPointer(3, GL_FLOAT, 0, NULL);

		glBindBuffer(GL_ARRAY_BUFFER, cboId);
		glBufferData(GL_ARRAY_BUFFER, 3 * n * sizeof(GLfloat), colors, GL_STATIC_DRAW);
		glColorPointer(3, GL_BYTE, 0, NULL);

		glPointSize(1.f);
		glDrawArrays(GL_POINTS, 0, n);

		glDisableClientState(GL_VERTEX_ARRAY);
		glDisableClientState(GL_COLOR_ARRAY);

		glfwSwapBuffers(window);
		glfwPollEvents();
	}
}

int render()
{
	GLFWwindow* window = initWindow(1024, 620);
	if (NULL != window)
	{
		display(window);
	}
	glfwDestroyWindow(window);
	glfwTerminate();
	return 0;
}

int main(int argc, char ** argv)
{
	render();
	return 0;
}